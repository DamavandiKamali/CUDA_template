
#include <hip/hip_runtime.h>
#include <stdio.h>
#define row1 20
#define col2 20
#define col1_row2 40
/////////////////////////////////////////////////////////
/// mul_matrix function ////////////////////////////
/////////////////////////////////////////////////////////
__global__ void mul_matrix(int *a,int *b,int *c) {
	int row = threadIdx.y;
	int col = threadIdx.x;
	int P_val = 0;
	for (int k = 0; k <col1_row2; ++k)
		P_val +=a[row + row1*k]*b[k + col*col1_row2 ] ;
	c[row+col*row1] = P_val;
	}
//////////////////////////////////////////////////////////
/// print_matrix function ///////////////////////////
////////////////////////////////////////////////////////
void print_matrix(int *c){
	for (int i = 0; i < row1; i++){
		for (int j = 0; j < col2; j++)
			printf("%4d",c[i+row1*j]);
		printf("\n\n");
	}
}
////////////////////////////////////////////////////////
/// main function ////////////////////////////////
////////////////////////////////////////////////////////
int main(){
	int a[row1*col1_row2];
	int b[col1_row2*col2];
	int c[row1*col2];
	for(int i=0;i<row1;i++)
		for(int j=0;j<col1_row2;j++)
			a[i+row1*j]=1;
	for(int i=0;i<col1_row2;i++)
		for(int j=0;j<col2;j++)
			b[i+col1_row2*j]=1;
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	hipEvent_t start,stop;
	int size1 = row1*col1_row2 * sizeof(int);
	int size2 = col1_row2*col2 * sizeof(int);
	int size3 = row1*col2 * sizeof(int);
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size1);
	hipMalloc((void **)&d_b, size2);
	hipMalloc((void **)&d_c, size3);
	// Copy inputs to device
	hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	dim3 dimGrid(1, 1);
	dim3 dimBlock(col2,row1);
	// Launch mul_matrix() kernel on GPU
	mul_matrix<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	// Copy result back to host
	hipMemcpy(c, d_c, size3, hipMemcpyDeviceToHost);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("time to generate:%3.5f ms\n",elapsedTime);
	//print_matrix(c);
	printf("c[%d]=%d\n",1,c[1]);
	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return(0);
}