#include "hip/hip_runtime.h"

/*
 * Driver APIC code that calls a runtime kernel
 * Vector addition: C = A + B.
 */

// Includes
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

// Variables
hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;
hipFunction_t vecAdd;
float* h_A;
float* h_B;
float* h_C;
hipDeviceptr_t d_A;
hipDeviceptr_t d_B;
hipDeviceptr_t d_C;

// Functions

__global__ void kernel(float* d_a, float* d_b, float* d_c, int n)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(idx < n)
    d_c[idx] = d_a[idx] + d_b[idx];
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i)
    data[i] = rand() / (float)RAND_MAX;
}

void errorExit()
{
  printf("Error exit!\n");
  exit(1);
}

// Host code
int main(int argc, char** argv)
{
  int N = 50000;
  unsigned int size = N * sizeof(float);
  hipError_t error;
  
  printf("Vector Addition (Driver API)\n");
  // Initialize
  error = hipInit(0);
  if (error != hipSuccess) errorExit();
  
  // Get number of devices supporting CUDA
  int deviceCount = 0;
  error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) errorExit();
  if (deviceCount == 0) {
    printf("There is no device supporting CUDA.\n");
    exit(1);
  }
  
  // Get handle for device 0
  error = hipDeviceGet(&cuDevice, 0);
  if (error != hipSuccess) errorExit();
  
  // Create context
  error = hipCtxCreate(&cuContext, 0, cuDevice);
  if (error != hipSuccess) errorExit();
  
  // Allocate input vectors h_A and h_B in host memory
  h_A = (float*)malloc(size);
  if (h_A == 0) errorExit();
  h_B = (float*)malloc(size);
  if (h_B == 0) errorExit();
  h_C = (float*)malloc(size);
  if (h_C == 0) errorExit();
  
  // Initialize input vectors
  RandomInit(h_A, N);
  RandomInit(h_B, N);
  
  // Allocate vectors in device memory
  error = hipMalloc(&d_A, size);
  if (error != hipSuccess) errorExit();
  error = hipMalloc(&d_B, size);
  if (error != hipSuccess) errorExit();
  error = hipMalloc(&d_C, size);
  if (error != hipSuccess) errorExit();
  
  // Copy vectors from host memory to device memory
  error = hipMemcpyHtoD(d_A, h_A, size);
  if (error != hipSuccess) errorExit();
  error = hipMemcpyHtoD(d_B, h_B, size);
  if (error != hipSuccess) errorExit();
  
  // Invoke kernel (Runtime API)
  int nThreadsPerBlk=128;
  int nBlks = (N/nThreadsPerBlk) + (((N%nThreadsPerBlk)>0)?1:0);
  kernel<<<nBlks,nThreadsPerBlk>>>((float*)d_A,(float*) d_B,(float*) d_C, N);
  
  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  error = hipMemcpyDtoH(h_C, d_C, size);
  if (error != hipSuccess) errorExit();
  
  // Verify result
  int i;
  for (i = 0; i < N; ++i) {
    float sum = h_A[i] + h_B[i];
    if (fabs(h_C[i] - sum) > 1e-7f) {
      printf("Mistake index %d %g %g\n",i,h_C[i],sum);
      break;
    }
  }
  printf("Test %s \n", (i == N) ? "PASSED" : "FAILED");
  return(0);
}
